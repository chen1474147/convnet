#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
/*
  Modified by Li Sijin, 2013  lisijin7@gmail.com
 */
#include <assert.h>

#include <layer_kernels.cuh>

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
                            const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp);
        
        /*
         * Compute the probability of guessing the correct case if you take the most-probable label.
         * 
         * This is done like this:
         * 
         * - If the most probable label is not equal to the true label, then the probability is zero.
         * - Otherwise, the probability is 1 / (number of labels whose probability is equal to the maximum).
         * 
         * This is certainly overkill -- in practice, it's just about impossible for two labels to get assigned
         * maximum probability. But it's a safety measure to prevent over-estimating your accuracy.
         * Though it could never happen in reality. Well it could. But it wouldn't. Cool?
         */
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}


/*
  Please don't use any special neuron (such as logistic, tanh) before this layer
  
  z_i is the input from the previous layer, y_i is the ground truth vectorized 1-of-K map
  x_i = logistic(z_i)
  what indlogpred calculate is y_i log( x_i) + (1-y_i) log(1-x_i)
  predmap:    (numTasks, numCases)
  indlogpred: (numTasks, numCases)
  correctprobs:(numTasks, numCases)

  each thread is responsible for per_thread_case position in one task
  blockIdx.x determines which task(indicator) to take
 */

__global__ void kEltwiseLogregCost(float* predmap, float* indmap, float*indlogpred, float* correctprobs, int numCases, int numTasks, int per_thread_case) {
  const int task_id = blockIdx.x;
  const int start_tx = threadIdx.x * per_thread_case;
  const int end_tx = min(start_tx + per_thread_case, numCases);
  if (task_id >= numTasks) {
    return;
  }
  for (int c_id = start_tx; c_id < end_tx; ++c_id) {
    int pos = task_id * numCases + c_id;
    float t = __fdividef(1.0f, 1.0f + __expf(-predmap[ pos ]));   
    if (indmap[pos] == 1) {
      indlogpred[pos] = __logf(t);
      correctprobs[pos] = t;
    } else {
      t = 1-t;
      indlogpred[pos] = __logf(t);
      correctprobs[pos] = t;
    }
  }
}

/*
  z_i is the input of previous layer
  x_i = logistic(z_i)
  Calculate the gradient of f(z_i) = y_i log x_i + (1-y_i) log(1-x_i)
  df_dz = [yi/xi + (yi-1)/(1-xi)]*(1-xi)(xi) = [yi * (1-xi) + (yi-1)*xi] = [yi - xi]
  predmap:       (numTasks, numCases)
  indmap:        (numCases, numCases)
  df_dz:    (numCases, numCases)

  each thread is responsible for per_thread_case cases in one task
  each block is responsible for one task  
 */
template <bool add>
__global__ void kEltwiseLogregGrad(float * predmap, float* indmap, float* df_dz, int numCases, int numTasks, int per_thread_case, float coeff ) {
  const int task_id = blockIdx.x;
  const int start_tx = threadIdx.x * per_thread_case;
  const int end_tx = min(start_tx + per_thread_case, numCases);
  if (task_id >= numTasks) {
    return;
  }
  for (int c_id = start_tx; c_id < end_tx; ++c_id) {
    int pos = task_id * numCases + c_id;
    float v = coeff * (indmap[pos] - __fdividef(1.0f, 1.0f + __expf(-predmap[ pos ])));   
    if (add) {
      df_dz[pos] += v;
    } else {
      df_dz[pos] = v;
    }
  }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dy_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregCostGrad(float* y_l, float* labels, float* dE_dy_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * (label == ty);
        v = __fdividef(v, y_l[tidx]);
        if (add) {
            dE_dy_l[tidx] += v;
        } else {
            dE_dy_l[tidx] = v;
        }
    }
}

/*
 * dE_dy_l: (numOut, numCases)
 * y_l:     (numOut, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kSoftmaxGrad(float* dE_dy_l, float* y_l, float* dE_dx_l, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        float v = 0;
        for (int j = 0; j < numOut; j++) {
            v += dE_dy_l[j * numCases + tx] * ((j == ty) - y_l[j * numCases + tx]);
        }
        v *= y_l[tidx];
        
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregSoftmaxGrad(float* y_l, float* labels, float* dE_dx_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * ((label == ty) - y_l[tidx]);
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

template <int B_X, bool add>
__global__ void kEltwiseMaxGrad(float* actGrad, float* input, float* output, float* target,
                                const int numElements) {
    for (int i = B_X * blockIdx.x + threadIdx.x; i < numElements; i += B_X * gridDim.x) {
        if (add) {
            target[i] += actGrad[i] * (output[i] == input[i]);
        } else {
            target[i] = actGrad[i] * (output[i] == input[i]);
        }
    }
}

void computeEltwiseMaxGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& output, NVMatrix& target, bool add) {
    assert(actGrad.isContiguous());
    assert(output.isContiguous());
    assert(input.isContiguous());
    assert(actGrad.isSameDims(input));
    assert(actGrad.isSameDims(output));
    
    dim3 blocks(DIVUP(actGrad.getNumElements(), 128));
    dim3 threads(128);
    if (add) {
        assert(actGrad.isSameDims(target));
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), true>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, true><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    } else {
        target.resize(actGrad);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), false>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, false><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    }
    
    getLastCudaError("computeEltwiseMaxGrad: Kernel execution failed");
}

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
void computeLogregCost(NVMatrix& labels, NVMatrix& probs, NVMatrix& labelLogProbs_out, NVMatrix& correctProbs_out) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    
    NVMatrix& maxProbs = probs.max(0);
    
    labelLogProbs_out.resize(1, numCases);
    correctProbs_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLogregCost), hipFuncCachePreferL1);
    kLogregCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     labelLogProbs_out.getDevData(), correctProbs_out.getDevData(),
                                     numCases, numOut);
    getLastCudaError("computeLogregCost: Kernel execution failed");
//    hipDeviceSynchronize();
    delete &maxProbs;
}

void computeLogregGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregCostGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregCostGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    getLastCudaError("computeLogregGrad: Kernel execution failed");
}

void computeSoftmaxGrad(NVMatrix& acts, NVMatrix& actsGrad, NVMatrix& target, bool add) {
    int numCases = acts.getLeadingDim();
    int numOut = acts.getFollowingDim();

    assert(acts.isSameDims(actsGrad));
    assert(acts.isContiguous());
    assert(actsGrad.isContiguous());
    assert(target.isContiguous());
    assert(acts.isTrans());
    assert(actsGrad.isTrans());

    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(acts);
        kSoftmaxGrad<false><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    } else {
        kSoftmaxGrad<true><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    }
    getLastCudaError("computeSoftmaxGrad: Kernel execution failed");
}

void computeLogregSoftmaxGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregSoftmaxGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregSoftmaxGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    getLastCudaError("computeLogregSoftmaxGrad: Kernel execution failed");
}


void computeEltwiseLogregCost(NVMatrix& indmap, NVMatrix& predmap, NVMatrix & indlogpred, NVMatrix& correctprobs) {
  int numCases = predmap.getNumCols();
  int numTasks = predmap.getNumRows();
  assert(indmap.getNumCols() == numCases);
  assert(indmap.getNumRows() == numTasks);
  assert(!indmap.isTrans());
  assert(!predmap.isTrans());
  assert(indmap.isContiguous());
  assert(predmap.isContiguous());

  indlogpred.resize(numTasks, numCases);
  correctprobs.resize(numTasks, numCases);
  dim3 threads(ELTLOGREG_ERR_THREADS_X, 1);
  dim3 blocks(numTasks, 1); // Ensure the numTasks will not exceed GPU's capacity
  int per_thread_case = DIVUP( numCases, ELTLOGREG_ERR_THREADS_X); 
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseLogregCost), hipFuncCachePreferL1);
  kEltwiseLogregCost<<<blocks, threads>>>(predmap.getDevData(), indmap.getDevData(), indlogpred.getDevData(), correctprobs.getDevData(), numCases, numTasks, per_thread_case);
  getLastCudaError("computeEltwiseLogregCost: Kernel execution failed");
}

void computeEltwiseLogregGrad(NVMatrix& indmap, NVMatrix& predmap, NVMatrix& target, bool add, float coeff) {
  int numCases = predmap.getLeadingDim();
  int numTasks = predmap.getFollowingDim();
  assert( indmap.getLeadingDim() == numCases);
  assert( indmap.getFollowingDim() == numTasks);
  assert(!indmap.isTrans());
  assert(!predmap.isTrans());
  assert(indmap.isContiguous());
  assert(predmap.isContiguous());

  dim3 threads(ELTLOGREG_ERR_THREADS_X, 1);
  dim3 blocks(numTasks, 1); // Ensure the numTasks will not exceed GPU's capacity
  int per_thread_case = DIVUP( numCases, ELTLOGREG_ERR_THREADS_X);
  if (!add) {
    target.resize(predmap);
    kEltwiseLogregGrad<false><<<blocks, threads>>>(predmap.getDevData(), indmap.getDevData(), target.getDevData(), numCases, numTasks, per_thread_case, coeff);
  } else {
    kEltwiseLogregGrad<true><<<blocks, threads>>>(predmap.getDevData(), indmap.getDevData(), target.getDevData(), numCases, numTasks, per_thread_case, coeff);
  }
  getLastCudaError("computeEltwiseLogregGrad: Kernel execution failed");
}
